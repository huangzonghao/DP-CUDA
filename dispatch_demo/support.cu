#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <fstream>

#include "timer.h"
#include "utils.h"

using namespace std;


#define SCALE 2.0
#define SHIFT 4.5
#define BLOCKS 1500
#define THREADS 1024
#define ARRAYSIZE 153600000


/* we are generating 1500 * 1024 * 100 = 153 600 000 random numbers on the device and then copy back to the host */

// set up the hiprandState for each individual thread
__global__ 
void setupSeeds(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    /* seed , sequence, offset, state */
    hiprand_init(7+id, id, 0, &state[id]);
}

// the kernel for TEST 1
// the internal looping mode
// each thread would have to loop 100 times
__global__
void test1(hiprandState *state, float *outputArray){
    int myID = threadIdx.x + blockIdx.x * blockDim.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[myID];
    /* Generate pseudo-random uniforms */

    for ( int i = 0; i < 100; ++i){
        outputArray[myID * 100 + i] = hiprand_normal(&localState);
    }

}




// the kernel for TEST 2
// the external looping mode
__global__ 
void test2(hiprandState *state, float *outputArray)
{
    int myID = threadIdx.x + blockIdx.x * blockDim.x;

    outputArray[myID] = hiprand_normal(&state[myID]);
}



void demo()
{


    hiprandState *d_curandStates;
    float *d_array, *h_array;
    int deviceIdx;
    struct hipDeviceProp_t deviceProperties;



    checkCudaErrors(hipGetDevice(&deviceIdx));
    checkCudaErrors(hipGetDeviceProperties(&deviceProperties,deviceIdx));



    h_array = (float *)calloc(ARRAYSIZE, sizeof(float));

    /* Allocate space for prng states on device */
    // a specific state for each thread
    checkCudaErrors(hipMalloc((void **)&d_curandStates, THREADS * BLOCKS * sizeof(hiprandState)));

    ofstream fs;




    GpuTimer timer1, timer2;


    /************** TEST 1 ************/
    /********** INTERNAL LOOPING ******/

    /* Setup prng states */
    setupSeeds<<<BLOCKS, THREADS>>>(d_curandStates);

    checkCudaErrors(hipMalloc(&d_array, ARRAYSIZE * sizeof(float)));  
    checkCudaErrors(hipMemset(d_array, 0, ARRAYSIZE * sizeof(float))); 

    timer1.Start();

    test1<<<BLOCKS, THREADS>>>(d_curandStates, d_array);
    checkCudaErrors(hipMemcpy(h_array, d_array, ARRAYSIZE * sizeof(float), hipMemcpyDeviceToHost));

    timer1.Stop();

    // fs.open("test1.txt");
    // for ( size_t i = 0; i < ARRAYSIZE; ++i){
    //     fs << h_array[i] << endl;
    // }
    // fs.close();

    checkCudaErrors(hipFree(d_array));
  



    /************** TEST 2 ************/
    /********** EXTERNEL LOOPING ******/

    // first reallocate the d_array
    checkCudaErrors(hipMalloc(&d_array, BLOCKS * THREADS * sizeof(float)));
    checkCudaErrors(hipMemset(d_array, 0, BLOCKS * THREADS * sizeof(float)));
    
    // refresh the hiprandState
    setupSeeds<<<BLOCKS, THREADS>>>(d_curandStates);



    timer2.Start();

    for (int i = 0; i < 100; ++i){

        test2<<<BLOCKS, THREADS>>>(d_curandStates, d_array);
        checkCudaErrors(hipMemcpy(h_array + i * 100, d_array, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost));

    }

    timer2.Stop();

    // fs.open("test2.txt");
    // for ( size_t i = 0; i < ARRAYSIZE; ++i){
    //     fs << h_array[i] << endl;
    // }
    // fs.close();

    int err = printf("TEST 1 ran in: %f msecs.\n", timer1.Elapsed());

    if (err < 0) {
      std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
      exit(1);
    }

    err = printf("TEST 2 ran in: %f msecs.\n", timer2.Elapsed());

    if (err < 0) {
      std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
      exit(1);
    }


    /* Cleanup */
    checkCudaErrors(hipFree(d_curandStates));
    checkCudaErrors(hipFree(d_array));
    free(h_array);

    return;
}