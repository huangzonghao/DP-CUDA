#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

// Hack: including source files is bad in general!
// But we know it is not going to be used anywhere else
// If you do, write a header file and include it instead!
#include "dp_model.cu"


// Helper function to get CUDA thread id
// whenever we use __device__ function
__device__ inline size_t
get_thread_id() {

    size_t blockId = blockIdx.x +
                     blockIdx.y * gridDim.x +
                     gridDim.x * gridDim.y * blockIdx.z;
    return blockId * blockDim.x + threadIdx.x;
}


// Using these values for general CUDA GPU is just fine
inline void
get_grid_dim(dim3* block_dim, dim3* grid_dim, size_t batch_size) {

    size_t n_block = batch_size / 512 + 1;

    assert(block_dim && grid_dim);
    *block_dim = dim3(512, 1, 1);
    *grid_dim = dim3(4096, n_block / 4096 + 1, 1);
}


// CUDA Kernel function for initialization
__global__ void
init_kernel(float *current_values,
            size_t batch_idx,
            size_t batch_size) {

    size_t thread_idx = get_thread_id();

    if (thread_idx < batch_size) {

        size_t current = batch_idx * batch_size + thread_idx;
        size_t parent = current - batch_size;

        if (current == 0) {
            current_values[current] = 0.0;
        } else {
            current_values[current] = current_values[parent] + 1.0;
        }
    }
}


// Plain C function for interact with kernel
void
init_states(float *current_values) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    // The very first state
    init_kernel<<<1, 1>>>(current_values, 0, 1);

    for (size_t d = 0; d < n_dimension; d++) {

        size_t batch_size = pow(n_capacity, d);

        dim3 block_dim, grid_dim;
        get_grid_dim(&block_dim, &grid_dim, batch_size);

        for (size_t batch_idx = 1; batch_idx < n_capacity; batch_idx++) {
            init_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 batch_idx,
                                                 batch_size);
        }
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();

}


// The CUDA kernel function for DP
__global__ void
iter_kernel(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values,
            int period,
            size_t batch_idx,
            size_t batch_size) {

    size_t thread_idx = get_thread_id();

    if (thread_idx < batch_size) {

        size_t current = batch_idx * batch_size + thread_idx;
        size_t parent = current - batch_size;

        // Custom filter
        if (current % n_capacity != 0) {
            current_values[current] = 0.0;
            return;
        }

        if (current == 0 || depletion[parent] == 0) {

            optimize(current_values,
                     // the state we are computing
                     current,
                     // n_depletion, min_depletion, max_depletion
                     depletion, 0, 2,
                     // n_order, min_order, max_order
                     order, 0, n_capacity,
                     // future utility for reference
                     future_values,
                     period);

        } else /* (depletion[parent] != 0) */ {

            optimize(current_values,
                     // the state we are computing
                     current,
                     // n_depletion, min_depletion, max_depletion
                     depletion, depletion[parent]+1, depletion[parent]+2,
                     // n_order, min_order, max_order
                     order, 0, n_capacity,
                     // future utility for reference
                     future_values,
                     period);

        }
    }
}


// Plain C function to interact with kernel
// The structure is essentially the same as init_states.
// If you feel confused, start from there!
void
iter_states(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values,
            int period) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    // The very first state 0,0,...,0
    iter_kernel<<<1, 1>>>(current_values,
                          depletion,
                          order,
                          future_values,
                          period,
                          0, 1);

    for (size_t d = 0; d < n_dimension; d++) {

        size_t batch_size = pow(n_capacity, d);

        dim3 block_dim, grid_dim;
        get_grid_dim(&block_dim, &grid_dim, batch_size);

        for (size_t batch_idx = 1; batch_idx < n_capacity; batch_idx++) {
            iter_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 depletion,
                                                 order,
                                                 future_values,
                                                 period,
                                                 batch_idx,
                                                 batch_size);
        }
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();

}
