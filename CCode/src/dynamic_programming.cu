#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include "dynamic_programming.h"

__device__ inline size_t
getGlobalIdx_3D_1D() {
    size_t blockId = blockIdx.x +
                     blockIdx.y * gridDim.x +
                     gridDim.x * gridDim.y * blockIdx.z;
    return blockId * blockDim.x + threadIdx.x;
}


// CUDA Kernel function for initialization
__global__ void
init_kernel(float *current_values,
            size_t d,
            size_t c,
            size_t batch_size) {

    size_t idx = getGlobalIdx_3D_1D();

    if (idx < batch_size) {
        size_t current = c * batch_size + idx;
        size_t parent = current - batch_size;

        current_values[current] = current_values[parent] + 1.0;
    }
}


// plain C function for interact with CUDA
void
init_states(float *current_values) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    for (size_t d = 0; d < n_dimension; d++) {
        size_t batch_size = pow(n_capacity, d);
        size_t n_thread = 512;
        size_t n_block = batch_size / n_thread + 1;

        dim3 block_dim(n_thread, 1, 1);
        dim3 grid_dim(4096, n_block / 4096 + 1, 1);

    for (unsigned d = 0; d < n_dimension; d++) {
        unsigned batch_size = pow(n_capacity, d);
        unsigned n_block = batch_size / n_thread + 1;
        for (size_t c = 1; c < n_capacity; c++) {
            init_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 d, c, batch_size);
        }
    }
}


__device__ inline int
sum(int *state, int length) {

    int acc = 0;
    for (int i = 0; i < length; i++) {
        acc += state[i];
    }
    return acc;
}


// Decode index and store the result into
// the state array by overwritting
// turning base 10 index into base ``n_capacity``
// without the last entry
// as it is always called with respect to today
__device__ inline void
decode(int *state, int index) {

    for (int i = n_dimension - 1; i >= 0; i--) {
        state[i] = index % n_capacity;
        index /= n_capacity;
    }
    state[n_dimension] = 0;
}


// The inverse function of ``decode``
// without the 0-th entry
// as it is always called with respect to future
__device__ inline int
encode(int *state) {

    int acc = 0;
    for (int i = 1; i < n_dimension + 1; i++) {
        acc *= n_capacity;
        acc += state[i];
    }
    return acc;
}


// The common component for both deplete and sell
// Original values in ``state`` are overwritten
__device__ inline int
substract(int *state, int length, int quantity) {

    int acc = 0;
    for (int i = 0; i < length; i++) {
        if (quantity <= state[i]) {
            acc += quantity;
            state[i] -= quantity;
            break;
        } else {
            quantity -= state[i];
            acc += state[i];
            state[i] = 0;
        }
    }
    return acc;
}


// Original values in ``state`` are overwritten
__device__ inline float
deplete(int *state, int quantity) {

    return unit_salvage * substract(state, n_dimension, quantity);
}


// Simple summation
__device__ inline float
hold(int *state) {

    return unit_hold * sum(state, n_dimension);
}


// Original values in ``state`` are overwritten
__device__ inline float
order(int *state, int quantity) {

    state[n_dimension] = quantity;
    return unit_order * quantity;
}


// Original values in ``state`` are overwritten
__device__ inline float
sell(int *state, int quantity) {

    return unit_price * substract(state, n_dimension+1, quantity);
}


// Original values in ``state`` are overwritten
__device__ inline float
dispose(int *state) {
    int disposal = state[0];
    state[0] = 0;
    return unit_disposal * disposal;
}


// Original values in ``state`` are overwritten
__device__ inline float
revenue(int *state,
        size_t current,
        int n_depletion,
        int n_order,
        int n_demand) {

    float depletion = deplete(state, n_depletion);
    float holding = hold(state);
    float ordering = order(state, n_order);
    float sales = sell(state, n_demand);
    float disposal = dispose(state);
    float revenue = depletion + holding + discount * (ordering + sales + disposal);

    return revenue;
}


__device__ void
optimize(float *current_values,
         size_t current,
         dp_int *depletion,
         int min_depletion,
         int max_depletion,
         dp_int *order,
         int min_order,
         int max_order,
         float *future_values) {

    // Allocate a memory buffer on stack
    // So we don't need to do it for every loop
    // The last dimension are used to store the ordering
    int state[n_dimension+1] = {};
    decode(state, current);

    int n_depletion = 0;
    int n_order = 0;
    float max_value = 0.0;

    for (int i = min_depletion; i < max_depletion; i++) {
        for (int j = min_order; j < max_order; j++) {

            float expected_value = 0.0;

            for (int k = min_demand; k < max_demand; k++) {

                // Initialize the ``state`` array
                // before each call of ``revenue()``
                decode(state, current);

                // By calling ``revenue()``, the state array
                // now stores the state for future
                float value = revenue(state, current, i, j, k);

                // And find the corresponding utility of future
                int future = encode(state);

                value += discount * future_values[future];

                expected_value += demand_distribution[k - min_demand] * value;
            }

            // Simply taking the moving maximum
            if (expected_value > max_value + 1e-6) {
                max_value = expected_value;
                n_depletion = i;
                n_order = j;
            }
        }
    }

    // Store the optimal point and value
    current_values[current] = max_value;
    depletion[current] = (dp_int) n_depletion;
    order[current] = (dp_int) n_order;
}


// The CUDA kernel function for DP
__global__ void
iter_kernel(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values,
            size_t d,
            size_t c,
            size_t batch_size) {

    size_t idx = getGlobalIdx_3D_1D();

    if (idx < batch_size) {

        size_t current = c * batch_size + idx;
        size_t parent = current - batch_size;

        if (depletion[parent] == 0) {

            optimize(current_values,
                     current,
                     // n_depletion: optimal point and range [min, max)
                     depletion,
                     0,
                     2,
                     // n_order: optimal point and range [min, max)
                     order,
                     0,
                     n_capacity,
                     // future utility for reference
                     future_values);

        } else /* (depletion[parent] != 0) */ {

            optimize(current_values,
                     current,
                     // n_depletion: optimal point and range [min, max)
                     depletion,
                     depletion[parent]+1,
                     depletion[parent]+2,
                     // n_order: optimal point and range [min, max)
                     order,
                     0,
                     n_capacity,
                     // future utility for reference
                     future_values);

        }

        __threadfence_system();
    }
}


// The plain C function to interact with CUDA
void
iter_states(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    for (size_t d = 0; d < n_dimension; d++) {
        size_t batch_size = pow(n_capacity, d);
        size_t n_thread = 512;
        size_t n_block = batch_size / n_thread + 1;

        dim3 block_dim(n_thread, 1, 1);
        dim3 grid_dim(4096, n_block / 4096 + 1, 1);

        for (size_t c = 1; c < n_capacity; c++) {
            iter_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 depletion,
                                                 order,
                                                 future_values,
                                                 d, c, batch_size);
        }
    }
}
