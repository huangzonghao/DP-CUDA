#include "hip/hip_runtime.h"
#include "support.h"
#include "parameters.h"
#include <iostream>


// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <memory>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
using namespace std;


extern size_t  valueTablesLength;

//the MSB is the number of items to be expired and the LSB is the number of the newly purchased items
/******** kernels ********/

/* power function */
__device__ inline
size_t ipow(size_t base, size_t exp)
{
    size_t result = 1;
    while (exp != 0)
    {
        if ((exp & 1) == 1)
            result *= base;
        exp >>= 1;
        base *= base;
    }
    return result;
}
/* convert the oneD index to m-d coordinate */
/* take mD as the base */
__device__ inline
void decode (size_t* mDIdx, size_t oneDIdx){

      for( size_t i = 0; i < m ; ++i){
            mDIdx[m - 1 - i] = oneDIdx % k;
            oneDIdx /= k;
      }

}

/* convert the mD coordinate to oneD index */
__device__ inline
void encode(size_t* mDIdx, size_t* oneDIdx){
        size_t result = 0;
        for (size_t i = 0; i < m; ++i){
               result += mDIdx[i] * ipow(k, m - 1 - i);     // can be optimized once set up a reference table for ipow(k,i)
        }

        *oneDIdx = result;
}

/* returns the total number of items stored */
__device__ inline
size_t checkStorage(size_t* mDarray){
    size_t result = 0;
    for (size_t i = 0; i < m ; ++i ){
            result += mDarray[i];
    }
    return result;
}

/* depleting */
/* note the MSB represents the number of items to be expired  */

__device__ inline
void depleteStorage( size_t* mDarray,  size_t d_amount){
        size_t buffer = 0;
        size_t i = 0;
        while(!d_amount && i < m){ 
                if ( !mDarray[i]){
                        ++i;
                        continue;
                }

                if(mDarray[i] >= d_amount )
                {
                        mDarray[i] -= d_amount;
                        d_amount  = 0;
                        break;
                }
                buffer = d_amount - mDarray[i];
                mDarray[i] = 0;
                d_amount = buffer;
                buffer = 0;
                ++i;
        }
}


// if you don't know how to do perfectly overloading, don't use the following shortcuts....
/* the data transmission between host and device, the host addr always come first */
void passToDevice(float* h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(d_array, h_array, length * sizeof(float), hipMemcpyHostToDevice));
        return;
}

/* since we are only deal with floating points in gpu, we may hard coded the data type to be float */
void passToDevice(const float* h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(d_array, h_array, length * sizeof(float), hipMemcpyHostToDevice));
        return;
}
void readFromDevice(float * h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(h_array, d_array, length * sizeof(float), hipMemcpyDeviceToHost));
        return;
}


/* Initialize the cuda device */
/* set the init value of all entries in the value table to 0 */
__global__ 
void kernel_deviceTableInit(float* d_valueTable, size_t arrayLength  ){    
  size_t stepSize = gridDim.x * blockDim.x;  // the total number of threads which have been assigned for this task
  size_t myStartIdx = blockDim.x * blockIdx.x + threadIdx.x;
  for (size_t i = myStartIdx; i < arrayLength; i += stepSize)
    d_valueTable[i] = 0;

  __syncthreads(); 

}
/* allocate the device memory and initialize the values (the data type is hard coded to float) */
void deviceTableInit(size_t numTables, float ** tables, size_t tableLengths, cudaInfoStruct * cudainfo){
       dim3 gridSize(cudainfo->numBlocks, 1, 1);
       dim3 blockSize(cudainfo->numThreadsPerBlock, 1, 1);

       for ( size_t i = 0; i < numTables; ++i){
               checkCudaErrors(hipMalloc(&tables[i], tableLengths * sizeof(size_t)));
               kernel_deviceTableInit<<< gridSize, blockSize>>>(tables[i], tableLengths);
       } 

       return;
}



/* evaluate the state value given z and q */
/* return the expected value over the demands */
// i don't have to get all the storage information to get the state value of today
__device__
float stateValue( size_t expiringToday, 
                  int storageToday, 
                  int z, int q, 
                  float * d_randomTable){

        float profit = 0;
        float sum = 0;
        for ( size_t i = min_demand; i < max_demand; ++i){
                profit = s * z                                                             // the money collected depletion 
                       - h * max(int(int(storageToday) - z) , 0)                                // the cost for holding all the items 
                       - alpha * c * q                                                       // the money spent on ordering new items
                       + alpha * r * min(int(i), int(storageToday - z + q))               // the total income from selling the products to the customers
                       - alpha * theta * max(int(expiringToday - z - i), 0);// the money spent on the expired items
                
                sum += profit * d_randomTable[i];
        }

        return sum;

}

/* use one d arrangement here */
__global__ 
void kernel_valueTableUpdateWithPolicy(  float* d_randomTable,
                                         float* d_valueTable,     // note both the value table and the temp table here hold the exact starting index for this kernel launch
                                         float* d_tempTable,
                                         size_t* d_mdidx,
                                         size_t depletionIndicator,
                                         size_t valueTablesLength,
                                         size_t batchIdx
                                         ){
  float bestresult = 0;
  //float bestq = 0;
  float tempresult = 0;
  size_t storageToday = 0;
  // this is both the thread index and the data index in this batch
  size_t myIdx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t dataIdx = myIdx + batchIdx * gridDim.x * blockDim.x;

  // size_t testnum = 1601;
  // if(dataIdx == testnum){
  //   printf(" \n Now printing the calculation of the entry \n %d \n", dataIdx);
  // }

  if(dataIdx < valueTablesLength){

          decode(&d_mdidx[myIdx * m], dataIdx);

          if(depletionIndicator){
                  storageToday = checkStorage(&d_mdidx[myIdx * m]);

                for ( size_t q = 0; q < k; ++q){
                      tempresult = stateValue( d_mdidx[myIdx * m], 
                                               storageToday, 
                                               depletionIndicator * T,  q, 
                                               d_randomTable
                                              );

                      if (tempresult > bestresult){
                        bestresult = tempresult;
                        //bestq = q;
                      }
            
                }

                d_tempTable[dataIdx] = bestresult;

          }
          else{
                // starting the brute force algorithm on q directly
                  storageToday = checkStorage(&d_mdidx[myIdx * m]);

                  // if( dataIdx == testnum){
                  //   printf("\n storage today : %d ", storageToday);
                  //   printf("\n expiring today : %d ", d_mdidx[myIdx * m]);
                  // }

                for ( size_t q = 0; q < k; ++q){
                      tempresult = stateValue( d_mdidx[myIdx * m], 
                                               storageToday, 
                                               0,  q, 
                                               d_randomTable
                                              );
                  // if( dataIdx == testnum){
                  //   printf("\n tempresult  <%d> : %f",q, tempresult);
                  // }
                  
                      if (tempresult > bestresult){
                        bestresult = tempresult;
                        //bestq = q;
                      }
            
                }
                  // if( dataIdx == testnum){
                  //   printf("\n");
                  // }
                d_tempTable[dataIdx] = bestresult; // the corresponding q stores in the bestq
          }
    }

}
/* update the value table for one day */
/* only need to hold 2 tables and update each one at a time */
void valueTableUpdateWithPolicy( float** d_valueTables, 
                                 size_t currentTableIdx, 
                                 size_t depletionIndicator,       // either zero or the expected demand for one day
                                 float * d_randomTable,
                                 cudaInfoStruct * cudainfo ){

  // each thread will take care of a state at once
  
  size_t * d_mdidx; 
  // assign to each thread some global memory to store the m D information
  checkCudaErrors(hipMalloc(&d_mdidx, cudainfo->numBlocks * cudainfo->numThreadsPerBlock * m * sizeof(size_t)));


  size_t batchAmount = valueTablesLength / cudainfo->numBlocks / cudainfo->numThreadsPerBlock + 1;

  for ( size_t i = 0; i < batchAmount; ++i){

    kernel_valueTableUpdateWithPolicy<<<cudainfo->numBlocks, cudainfo->numThreadsPerBlock>>>(d_randomTable, d_valueTables[1 - currentTableIdx], d_valueTables[currentTableIdx], d_mdidx,depletionIndicator, valueTablesLength, i);
  }


  checkCudaErrors(hipFree(d_mdidx));
  return;
}


/* write in the values of the last day in the period */

// note the shared memory is shared among all the threads and has a limit per block
__global__
void kernel_presetValueTable(float * d_valueTable, size_t * d_mdidx, size_t table_length){

  size_t stepSize = gridDim.x * blockDim.x;  // the total number of threads which have been assigned for this task, oneD layout everywhere
  size_t myStartIdx = blockIdx.x * blockDim.x + threadIdx.x;

  for (size_t i = myStartIdx; i < table_length; i += stepSize){
    decode(&d_mdidx[myStartIdx * m], i);
    d_valueTable[i] = checkStorage(&d_mdidx[myStartIdx * m]) * s;
  }
  __syncthreads(); 
}

/* the interface for the main function */
void presetValueTable(float * d_valueTable, size_t  table_length, cudaInfoStruct * cudainfo){
  dim3 gridSize(cudainfo->numBlocks, 1, 1);
  dim3 blockSize(cudainfo->numThreadsPerBlock, 1, 1);
  size_t * d_mdidx; 
  // assign to each thread some global memory to store the m D information
  checkCudaErrors(hipMalloc(&d_mdidx, cudainfo->numBlocks * cudainfo->numThreadsPerBlock * m * sizeof(size_t)));
  kernel_presetValueTable<<<gridSize, blockSize>>> ( d_valueTable, d_mdidx, table_length);
  checkCudaErrors(hipFree(d_mdidx));
  return;
}

/* Gather the system information, for auto fill in the block number and the thread number per block */
void gatherSystemInfo(cudaInfoStruct * cudainfo){

  hipGetDeviceCount((int*)&(cudainfo->deviceCount));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  (cudainfo->numThreadsPerBlock) = deviceProp.maxThreadsPerBlock;
  (cudainfo->numBlocks) = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;

  return;

}



/*************** testing functions ********************/
__global__
void kernel_test(size_t i, size_t * list){
  size_t num = 0;
  if ( threadIdx.x == 1){

      encode(list, &num);
      printf("the result from kernel one is : %d \n", num);
      printf(" now printing the d_list with number m : %d \n",m);
      for (int i = 0; i < m; ++i){
        printf("<%d>", list[i]);
      }
      printf("\n");
  }
}
void test(){

  size_t  ** h_list;
  h_list = (size_t **)malloc(2 * sizeof(size_t *));
  h_list[0] = (size_t * )malloc(20 * sizeof(size_t));
  h_list[1] = (size_t * )malloc(20 * sizeof(size_t));
  // float ** h_list;
  // h_list = (float **)malloc(2 * sizeof(float *));
  // h_list[0] = (float * )malloc(20 * sizeof(float));
  // h_list[1] = (float * )malloc(20 * sizeof(float));

  for (int i = 0; i < m; ++i){
    h_list[0][i] = i * 2 + 1;
  }
  size_t * d_list;
  checkCudaErrors(hipMalloc(&d_list, m * sizeof(size_t)));

   // passToDevice(h_list[0], d_list, m);
  checkCudaErrors(hipMemcpy(d_list, h_list[0], m * sizeof(size_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( h_list[1], d_list, m * sizeof(size_t), hipMemcpyDeviceToHost));

   // readFromDevice(h_list[1], d_list, m);
   cout << endl << "this is the h_list1" << endl;
   for (int i = 0; i < m ; ++i){
     cout << h_list[0][i] << " ";
   }
      cout << endl << "this is the h_list2" << endl;
   for (int i = 0; i < m ; ++i){
     cout << h_list[1][i] << " ";
   }
   cout << endl;

  kernel_test<<<1, 1024>>>(77, (size_t* )d_list);


// test for one d to m d 

  return;
}



