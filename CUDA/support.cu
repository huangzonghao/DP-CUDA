#include "hip/hip_runtime.h"
#include "support.h"
#include "parameters.h"

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern unsigned long  valueTablesLength;

//the MSB is the number of items to be expired and the LSB is the number of the newly purchased items
/******** kernels ********/

/* power function */
__device__ inline
long ipow(size_t base, size_t exp)
{
    long result = 1;
    while (exp != 0)
    {
        if ((exp & 1) == 1)
            result *= base;
        exp >>= 1;
        base *= base;
    }
    return result;
}
/* convert the oneD index to m-d coordinate */
__device__ inline
void oneDtomD (long oneDIdx, size_t* mDIdx){

      for( size_t i = 0; i < m ; ++i){
            mDIdx[i] = oneDIdx % k;
            oneDIdx /= k;
      }

}

/* convert the mD coordinate to oneD index */
__device__ inline
void mDtooneD(size_t* mDIdx, long &oneDIdx){
        long result = 0;
        for (size_t i = 0; i < m; ++i){
               result += mDIdx[i] * ipow(k, i);     // can be optimized once set up a reference table for ipow(k,i)
        }

        oneDIdx = result;
}

/* returns the total number of items stored */
__device__ inline
size_t checkStorage(size_t* mDarray){
      size_t result = 0;
     for (size_t i = 0; i < m ; ++i ){
             result += mDarray[i];
     }
     return result;
}

/* depleting */
/* note the MSB represents the number of items to be expired  */

__device__ inline
void depleteStorage( size_t* mDarray,  size_t d_amount){
        size_t buffer = 0;
        size_t i = 0;
        while(!d_amount && i < m){ 
                if ( !mDarray[i]){
                        ++i;
                        continue;
                }

                if(mDarray[i] - d_amount >= 0)
                {
                        mDarray[i] -= d_amount;
                        d_amount  = 0;
                        break;
                }
                buffer = d_amount - mDarray[i];
                mDarray[i] = 0;
                d_amount = buffer;
                buffer = 0;
                ++i;
        }
}

/* the data transmission between host and device, the host addr always come first */
void passToDevice(float* h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(d_array, h_array, length, hipMemcpyHostToDevice));
        return;
}
void readFromDevice(float * h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(h_array, d_array, length, hipMemcpyDeviceToHost));
        return;
}

/* Initialize the cuda device */
/* set the init value of all entries in the value table to 0 */
__global__ 
void kernel_deviceTableInit(float* d_valueTable, long arrayLength  ){    
  long stepSize = gridDim.x * gridDim.y * blockDim.x * blockDim.y;  // the total number of threads which have been assigned for this task
  long myStartIdx = (gridDim.x * blockIdx.y + blockIdx.x - 1) * blockDim.x * blockDim.y +  threadIdx.y * blockDim.x + threadIdx.x;
  for (long long i = myStartIdx; i < arrayLength; i += stepSize)
    d_valueTable[i] = 0;

  __syncthreads(); 

}
/* allocate the device memory and initialize the values (the data type is hard coded to float)*/
void deviceTableInit(size_t numTables, float ** tables, unsigned long tableLengths, cudaInfoStruct * cudainfo){
       dim3 gridSize(cudainfo->numBlocks, 1, 1);
       dim3 blockSize(cudainfo->numThreadsPerBlock, 1, 1);

       for ( size_t i = 0; i < numTables; ++i){
               checkCudaErrors(hipMalloc(tables[i], tableLengths));
               deviceTableInitKernel<<< gridSize, blockSize>>>(tables[i], tableLengths[i]);
       } 

       return;
}



/* evaluate the state value given z and q */
/* return th expected value over the demands */
// to complicit to be an inline function
__device__
float stateValue( size_t expiringToday, 
                  size_t storageToday, 
                  size_t z, size_t q, 
                  size_t * d_randomTable, 
                  size_t min_demand, size_t max_demand){

         float profit = 0;
         float sum = 0;
         for ( size_t i = 0; i < numDemands; ++i){
                 profit = s * z                                                             // the money collected depletion 
                        - h * max(int(storageToday) - z , 0)                                // the cost for holding all the items 
                        - alpha * c * q                                                       // the money spent on ordering new items
                        + alpha * r * min(i, storageToday - z + q)               // the total income from selling the products to the customers
                        - alpha * theta * max(expiringToday - z - i, 0);// the money spent on the expired items
                 sum += profit * d_randomTable[i];
         }

         return sum;
}

/* use one d arrangement here */
__global__ 
void kernel_valueTableUpdateWithPolicy(  float* d_randomTable,
                                         float* d_valueTable,     // note both the value table and the temp table here hold the exact starting index for this kernel launch
                                         float* d_tempTable,
                                         size_t depletionIndicator,
                                         size_t batchIdx
                                         ){
  // declare the local variables 
  extern __shared__ size_t mDIdx[];
  float bestresult = 0;
  float bestq = 0;
  float tempresult = 0;
  size_t storageToday = 0;
  // this is both the thread index and the data index in this batch
  size_t myIdx = blockIdx.x * gridDim.x + threadIdx.x;
  size_t dataIdx = myIdx + batchIdx * gridDim.x * blockDim.x;

  oneDtomD(dataIdx,mDIdx);

  if(depletionIndicator){

  }
  else{
        // starting the brute force algorithm on q directly
        for ( size_t q = 0; q < k + 1; ++q){
              storageToday = checkStorage(mDIdx);
              tempresult = stateValue( mDIdx[0], 
                                       storageToday, 
                                       0,  q, 
                                       d_randomTable, 
                                       min_demand, max_demand);
          
              if (tempresult > bestresult){
                bestresult = tempresult;
                bestq = q;
              }
    
        }
        d_tempTable[dataIdx] = bestresult; // the corresponding q stores in the bestq

  }

}
/* update the value table for one day */
/* only need to hold 2 tables and update each one at a time */
void valueTableUpdateWithPolicy( float** d_valueTables, 
                                 size_t currentTableIdx, 
                                 size_t depletionIndicator,       // either zero or the expected demand for one day
                                 float * d_randomTable,
                                 cudaInfoStruct cudainfo ){
// each thread will take care of a state at once
  size_t batchAmount = valueTablesLength / cudainfo->numBlocks / cuda->numThreadsPerBlock + 1;
  unsigned long cursor = 0; // holding the index of the next element to be sent to the kernel
  for ( size_t i = 0; i < batchAmount; ++i){
    kernel_valueTableUpdateWithPolicy<<<cudainfo->numBlocks, cudainfo->numThreadsPerBlock>>>(d_randomTable, 
                                                                                              d_valueTables[1 - currentTableIdx], 
                                                                                              d_valueTables[currentTableIdx], 
                                                                                              depletionIndicator,
                                                                                              i);
    cursor += cudainfo->numBlocks * cudainfo->numThreadsPerBlock;
  }

  return;

}


/* write in the values of the last day in the period */

__global__
void kernel_presetValueTable(float * d_valueTable, long long table_length){
  extern __shared__ size_t mDIdx[];
  long stepSize = gridDim.x * blockDim.x;  // the total number of threads which have been assigned for this task, oneD layout everywhere
  long myStartIdx = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned long i = myStartIdx; i < table_length; i += stepSize){
    oneDtomD(i,mDIdx);
    d_valueTable[i] = checkStorage(mDIdx) * s;
  }

  __syncthreads(); 
}

/* the interface for the main function */
void presetValueTable(float * d_valueTable, unsigned long  table_length, cudaInfoStruct * cudainfo){
  kernel_presetValueTable<<<cudainfo->numBlocks, cudainfo->numThreadsPerBlock, m * sizeof(size_t) >>>(float * d_valueTable, size_t table_length);
  return;
}

/* Gather the system information, fol auto fill in the block number and the thread number per block */
void gatherSystemInfo(size_t * deviceCount, size_t * numBlocks, size_t * numThreadsPerBlock){
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  hipGetDeviceCount(deviceCount);

  *numThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  numBlocks = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;
  return;

}
