#include "hip/hip_runtime.h"
#include "support.h"
#include "parameters.h"
#include "model.h"
#include <iostream>


// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <memory>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
using namespace std;


extern size_t  valueTablesLength;

//the MSB is the number of items to be expired and the LSB is the number of the newly purchased items
/******** kernels ********/

/* power function */
__device__ inline
size_t ipow(size_t base, size_t exp)
{
    size_t result = 1;
    while (exp != 0)
    {
        if ((exp & 1) == 1)
            result *= base;
        exp >>= 1;
        base *= base;
    }
    return result;
}
/* convert the oneD index to m-d coordinate */
/* take mD as the base */
__device__ inline
void decode (size_t* mDIdx, size_t oneDIdx){

      for( size_t i = 0; i < m ; ++i){
            mDIdx[m - 1 - i] = oneDIdx % k;
            oneDIdx /= k;
      }

}

/* convert the mD coordinate to oneD index */
__device__ inline
void encode(size_t* mDIdx, size_t* oneDIdx){
        size_t result = 0;
        for (size_t i = 0; i < m; ++i){
               result += mDIdx[i] * ipow(k, m - 1 - i);     // can be optimized once set up a reference table for ipow(k,i)
        }

        *oneDIdx = result;
}

/* returns the total number of items stored */
__device__ inline
size_t checkStorage(size_t* mDarray){
    size_t result = 0;
    for (size_t i = 0; i < m ; ++i ){
            result += mDarray[i];
    }
    return result;
}

/* depleting */
/* note the MSB represents the number of items to be expired  */

__device__ inline
void depleteStorage( size_t* mDarray,  size_t d_amount){
        size_t buffer = 0;
        size_t i = 0;
        while(!d_amount && i < m){ 
                if ( !mDarray[i]){
                        ++i;
                        continue;
                }

                if(mDarray[i] >= d_amount )
                {
                        mDarray[i] -= d_amount;
                        d_amount  = 0;
                        break;
                }
                buffer = d_amount - mDarray[i];
                mDarray[i] = 0;
                d_amount = buffer;
                buffer = 0;
                ++i;
        }
}


// if you don't know how to do perfectly overloading, don't use the following shortcuts....
/* the data transmission between host and device, the host addr always come first */
void passToDevice(float* h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(d_array, h_array, length * sizeof(float), hipMemcpyHostToDevice));
        return;
}

/* since we are only deal with floating points in gpu, we may hard coded the data type to be float */
void passToDevice(const float* h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(d_array, h_array, length * sizeof(float), hipMemcpyHostToDevice));
        return;
}
void readFromDevice(float * h_array, float* d_array, size_t length){
        checkCudaErrors(hipMemcpy(h_array, d_array, length * sizeof(float), hipMemcpyDeviceToHost));
        return;
}


/* Initialize the cuda device */
/* set the init value of all entries in the value table to 0 */
__global__ 
void kernel_deviceTableInit(float* d_valueTable, size_t arrayLength  ){    
  size_t stepSize = gridDim.x * blockDim.x;  // the total number of threads which have been assigned for this task
  size_t myStartIdx = blockDim.x * blockIdx.x + threadIdx.x;
  for (size_t i = myStartIdx; i < arrayLength; i += stepSize)
    d_valueTable[i] = 0;

  __syncthreads(); 

}
/* allocate the device memory and initialize the values (the data type is hard coded to float) */
void deviceTableInit(size_t numTables, float ** tables, size_t tableLengths, cudaInfoStruct * cudainfo){
       dim3 gridSize(cudainfo->numBlocks, 1, 1);
       dim3 blockSize(cudainfo->numThreadsPerBlock, 1, 1);

       for ( size_t i = 0; i < numTables; ++i){
               checkCudaErrors(hipMalloc(&tables[i], tableLengths * sizeof(size_t)));
               kernel_deviceTableInit<<< gridSize, blockSize>>>(tables[i], tableLengths);
       } 

       return;
}





/* Gather the system information, for auto fill in the block number and the thread number per block */
void gatherSystemInfo(cudaInfoStruct * cudainfo){

  hipGetDeviceCount((int*)&(cudainfo->deviceCount));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  (cudainfo->numThreadsPerBlock) = deviceProp.maxThreadsPerBlock;
  (cudainfo->numBlocks) = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;

  return;

}



/*************** testing functions ********************/
__global__
void kernel_test(size_t i, size_t * list){
  size_t num = 0;
  if ( threadIdx.x == 1){

      encode(list, &num);
      printf("the result from kernel one is : %d \n", num);
      printf(" now printing the d_list with number m : %d \n",m);
      for (int i = 0; i < m; ++i){
        printf("<%d>", list[i]);
      }
      printf("\n");
  }
}
void test(){

  size_t  ** h_list;
  h_list = (size_t **)malloc(2 * sizeof(size_t *));
  h_list[0] = (size_t * )malloc(20 * sizeof(size_t));
  h_list[1] = (size_t * )malloc(20 * sizeof(size_t));
  // float ** h_list;
  // h_list = (float **)malloc(2 * sizeof(float *));
  // h_list[0] = (float * )malloc(20 * sizeof(float));
  // h_list[1] = (float * )malloc(20 * sizeof(float));

  for (int i = 0; i < m; ++i){
    h_list[0][i] = i * 2 + 1;
  }
  size_t * d_list;
  checkCudaErrors(hipMalloc(&d_list, m * sizeof(size_t)));

   // passToDevice(h_list[0], d_list, m);
  checkCudaErrors(hipMemcpy(d_list, h_list[0], m * sizeof(size_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( h_list[1], d_list, m * sizeof(size_t), hipMemcpyDeviceToHost));

   // readFromDevice(h_list[1], d_list, m);
   cout << endl << "this is the h_list1" << endl;
   for (int i = 0; i < m ; ++i){
     cout << h_list[0][i] << " ";
   }
      cout << endl << "this is the h_list2" << endl;
   for (int i = 0; i < m ; ++i){
     cout << h_list[1][i] << " ";
   }
   cout << endl;

  kernel_test<<<1, 1024>>>(77, (size_t* )d_list);


// test for one d to m d 

  return;
}



